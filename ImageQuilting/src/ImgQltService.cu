#include "hip/hip_runtime.h"
/*
 * ImgQltService.cu
 *
 *  Created on: 07-Oct-2015
 *      Author: mobin
 */

#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include"opencv2/highgui/highgui.hpp"
#include<hip/hip_runtime.h>
#include<time.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include"opencv2/cudaarithm.hpp"
#include "tbb/tbb_stddef.h"

#define SAMPLE_SIZE 20
#define OVERLAP_SIZE 5

using std::cout;
using std::endl;
using namespace cv::cuda;

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number) {
	if(err!=hipSuccess) {
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

int outputX_size = 250;
int outputY_size = 250;
int sample_size = 20;
int overlap_size = 5;

typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

__device__ uchar* getSubImg(uchar* dSrc, int row, int col, int step) {
	return &dSrc[step * row + col * 3];
}

__device__ uchar getGrayElement(uchar* subArray, int row, int col, int step) {
	int b = subArray[row * step + col];
	int g = subArray[row * step + col + 1];
	int r = subArray[row * step + col + 2];
	return 0.2989 * r + 0.5870 * g + 0.1140 * b;
}

__device__ uchar getElement(uchar* subArray, int row, int col, int step) {
	return subArray[row * step + col];
}

__global__ void cudaGetPatch(uchar* dSrc, uchar* currImg, int step) {
	int blkcolIdx = blockIdx.x;
	int blkrowIdx = blockIdx.y;

	int colIdx = threadIdx.x;
	int rowIdx = threadIdx.y;

	uchar* subArray = getSubImg(dSrc, blkrowIdx, blkcolIdx, step);
	__shared__ uchar subImg[SAMPLE_SIZE][SAMPLE_SIZE*3];
	subImg[rowIdx][colIdx] = getElement(subArray, rowIdx, colIdx, step);


}

__global__ void cudaGetMinSSDImg(uchar* dSrc, uchar* preImg, uchar* topImg, int step, float* ssidArr) {

	int blkcolIdx = blockIdx.x;
	int blkrowIdx = blockIdx.y;

	int colIdx = threadIdx.x;
	int rowIdx = threadIdx.y;

	uchar* subArray = getSubImg(dSrc, blkrowIdx, blkcolIdx, step);

	__shared__ uchar subImgGray[SAMPLE_SIZE][SAMPLE_SIZE];
	__shared__ uchar preImgGray[SAMPLE_SIZE][SAMPLE_SIZE];
	__shared__ uchar topImgGray[SAMPLE_SIZE][SAMPLE_SIZE];

	subImgGray[rowIdx][colIdx] = getGrayElement(subArray, rowIdx, colIdx * 3, step);
	//printf("%u", subImgGray[rowIdx][colIdx]);
	if (preImg != 0) {
		preImgGray[rowIdx][colIdx] = getGrayElement(preImg, rowIdx, colIdx * 3, step);
		//printf("%u", preImgGray[rowIdx][colIdx]);
	}
	if (topImg != 0) {
		topImgGray[rowIdx][colIdx] = getGrayElement(topImg, rowIdx, colIdx * 3, step);
		//printf("%u", topImgGray[rowIdx][colIdx]);
	}

	__syncthreads();

	if (rowIdx == 0 && colIdx == 0) {
		int ssid = 0;

		if (preImg != 0) {
			for(int i = 0; i < SAMPLE_SIZE; i++) {
				for(int j = 0; j < OVERLAP_SIZE; j++) {
					int diff = subImgGray[i][j] - preImgGray[i][SAMPLE_SIZE - OVERLAP_SIZE + i];
					ssid += sqrtf((float) (diff * diff));
				}
			}
		}

		if (topImg != 0) {
			for(int i = 0; i < OVERLAP_SIZE; i++) {
				for(int j = 0; j < SAMPLE_SIZE; j++) {
					int diff = subImgGray[i][j] - topImgGray[SAMPLE_SIZE - OVERLAP_SIZE + i][j];
					ssid += sqrtf((float) (diff * diff));
				}
			}
		}
		ssidArr[(blkrowIdx * gridDim.y) + blkcolIdx] = ssid;
	}
}

__global__ void copyImg(uchar* dSrc, uchar* dDst, int height, int width, int sample_size, int step) {

	int colIdx = blockIdx.x + threadIdx.x;
	int rowIdx = blockIdx.y + threadIdx.y;

	printf("\nhello world again");

	dDst[(step*rowIdx+3*colIdx)]=dSrc[(step*rowIdx+3*colIdx)];
	dDst[(step*rowIdx+3*colIdx)+1]=dSrc[(step*rowIdx+3*colIdx)+1];
	dDst[(step*rowIdx+3*colIdx)+2]=dSrc[(step*rowIdx+3*colIdx)+2];
	//dDst[(xIndex * height) + yIndex] = dSrc;//dSrc(cv::Range(i, i + sample_size), cv::Range(j, j + sample_size));
}

std::vector<cv::Mat> createImageList(cv::Mat& hSrc) {
	int height = hSrc.rows;
	int width = hSrc.cols;

	std::vector<cv::Mat> imglist((height - sample_size) * (width - sample_size));
	for(int i = 0; i < height - sample_size; i++) {
		for(int j = 0; j < width - sample_size; j++) {
			imglist[(i * (width - sample_size)) + j] = hSrc(cv::Range(i, i + sample_size), cv::Range(j, j + sample_size));
		}
	}

	cv::cuda::GpuMat dDst(height, width, CV_8UC3);

	return imglist;
}

double getPixelValue(cv::Vec3b& pixel) {
	int b = pixel[0];
	int g = pixel[1];
	int r = pixel[2];

	//cout << "b " << b << "g " << g << "r "<< r <<  endl;
	//cout << "result:" << 0.2989 * r + 0.5870 * g + 0.1140 * b << endl;
	return 0.2989 * r + 0.5870 * g + 0.1140 * b;
}

int computeSSD(cv::Mat& overlap1, cv::Mat& overlap2) {
	double sum = 0;
	for (int i = 0; i < overlap1.rows; i++) {
		for (int j = 0; j < overlap1.cols; j++) {
			double val1 = getPixelValue(overlap1.at<cv::Vec3b>(i,j));
			//cout << "val1: " << val1 << endl;
			double val2 = getPixelValue(overlap2.at<cv::Vec3b>(i,j));
			//cout << "val2: " << val2 << endl;
			sum += std::sqrt(std::pow((val1 - val2), 2 ));
			//cout << "sum " << sum << endl;
		}
		//	sum += ((oi1[j] - oi2[j]) ^ 2) ^ 0.5;
	}

	//cout << "overlap2.rows " << overlap1.rows << endl;
	//cout << "overlap2.cols " << overlap1.cols << endl;

	return sum;
}

int computeHorizontalSSD(cv::Mat& topImg, cv::Mat& randImg, int overlap_size) {
	if(topImg.dims == 0) {
		return 0;
	}
	cv::Mat overlap1 = topImg(cv::Range(topImg.rows-overlap_size, topImg.rows), cv::Range(0,topImg.cols));
	cv::Mat overlap2 = randImg(cv::Range(0, overlap_size), cv::Range(0,topImg.cols));
	return computeSSD(overlap1, overlap2);
}

int computeVerticalSSD(cv::Mat& prevImg, cv::Mat& randImg, int overlap_size) {
	if(prevImg.dims == 0) {
		return 0;
	}
	cv::Mat overlap1 = prevImg(cv::Range(0,prevImg.rows),cv::Range(prevImg.cols-overlap_size,prevImg.cols));
	cv::Mat overlap2 = randImg(cv::Range(0,randImg.rows),cv::Range(0,overlap_size));
	return computeSSD(overlap1, overlap2);
}

int computeCombinedSSD(cv::Mat& prevImg, cv::Mat& topImg, cv::Mat& randImg, int overlap_size) {
	double verticalSSD = computeVerticalSSD(prevImg, randImg, overlap_size);
	double horizontalSSD = computeHorizontalSSD(topImg, randImg, overlap_size);
	return verticalSSD + horizontalSSD;
}

cv::Mat getMinSSDImg(cv::Mat& prevImg, cv::Mat& topImg, cv::Mat& hSrc, int width, int height) {
	/*int minSSD = 0;
	int minIdx = 0;
	for(int i = 0; i < imglist.size(); i++) {
		if(i == 0) {
			minSSD = computeCombinedSSD(prevImg, topImg, imglist[i], overlap_size);
			minIdx = i;
		} else {
			int ssd = computeCombinedSSD(prevImg, topImg, imglist[i], overlap_size);
			if(ssd < minSSD) {
				minSSD = ssd;
				minIdx = i;
			}
		}
	}*/

	cv::cuda::GpuMat dSrc, d_prevImg, d_topImg;
	dSrc.upload(hSrc);
	d_prevImg.upload(prevImg);
	d_topImg.upload(topImg);

	cv::cuda::GpuMat d_curImg(SAMPLE_SIZE, SAMPLE_SIZE, CV_8UC3);
	const dim3 grid(width-sample_size,height-sample_size);
	const dim3 block(sample_size,sample_size);

	float h_ssidArr[width-SAMPLE_SIZE][height-SAMPLE_SIZE];
	float* d_ssidArr;
	size_t arraysize = (width - SAMPLE_SIZE) * (height - SAMPLE_SIZE) * sizeof(float);


	SAFE_CALL(hipMalloc<float>(&d_ssidArr,arraysize),"CUDA Malloc Failed");

	cudaGetMinSSDImg<<<grid,block>>>(dSrc.ptr(), d_prevImg.ptr(), d_topImg.ptr(), dSrc.step, d_ssidArr);
	hipDeviceSynchronize();

	SAFE_CALL(hipMemcpy(h_ssidArr,d_ssidArr,arraysize,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	float minssid = FLT_MAX;
	int rowidx = 0;
	int colidx = 0;
	for(int i = 0; i < height - SAMPLE_SIZE; i++) {
		//printf("\n");
		for(int j = 0; j < width - SAMPLE_SIZE; j++) {
			//printf("\t%f",h_ssidArr[i][j]);
			if(minssid > h_ssidArr[i][j]) {
				minssid = h_ssidArr[i][j];
				rowidx = i;
				colidx = j;
			}
		}
	}

	printf("minssid : %f",minssid);
	cv::Mat curImg;
	d_curImg.download(curImg);
	return curImg;
}

cv::Mat getPreviousImg(int i, int j, cv::Mat& hDst) {
	cv::Mat subImg;
	if(j == 0) {
		return subImg;
	} else {
		subImg = hDst(cv::Range((i*sample_size)-(overlap_size*i),((i+1)*sample_size)-(overlap_size*i)),cv::Range(((j-1)*sample_size)-(j-1)*overlap_size,(j*sample_size)-(j-1)*overlap_size));
		return subImg;
	}
}

cv::Mat getTopImg(int i, int j, cv::Mat& hDst) {
	cv::Mat subImg;
	if(i == 0) {
		return subImg;
	} else {
		subImg = hDst(cv::Range(((i-1)*sample_size)-(i-1)*overlap_size,(i*sample_size)-(i-1)*overlap_size),cv::Range((j*sample_size)-(overlap_size*j),((j+1)*sample_size)-(overlap_size*j)));
		return subImg;
	}
}

void placeImg(int row, int col, cv::Mat& tile, cv::Mat& lImg) {
	int x1 = (row*sample_size)-(overlap_size*row);
	int x2 = ((row+1)*sample_size)-(overlap_size*row);
	int y1 = (col*sample_size)-(overlap_size*col);
	int y2 = ((col+1)*sample_size)-(overlap_size*col);
	if(row == 0) {
		x1 = (row*sample_size);
		x2 = ((row+1)*sample_size);
	}
	if(col == 0){
		y1 = (col*sample_size);
		y2 = ((col+1)*sample_size);
	}

	tile.copyTo(lImg(cv::Range(x1, x2), cv::Range(y1, y2)));
}

void imageQuilting(cv::Mat& hSrc, cv::Mat& hDst) {

	int height = hSrc.rows;
	int width = hSrc.cols;
	//std::cout << "inside image quilting" << endl;

	cv::cuda::GpuMat dDst;

	std::vector<cv::Mat> imglist = createImageList(hSrc);

	int nx = outputX_size/(sample_size - overlap_size);
	int ny = outputY_size/(sample_size - overlap_size);
	int newx = nx + (height - nx * overlap_size) / sample_size;
	int newy = ny + (width - ny * overlap_size) / sample_size;

	for(int i = 0; i < newx; i++ ) {
		for(int j = 0; j < newy; j++) {
			//cout << "i , j : " << i << " : " << j << endl;

			cv::Mat prevImg = getPreviousImg(i, j, hDst);

			cv::Mat topImg = getTopImg(i, j, hDst);

			cv::Mat currImg;
			if(i == 0 && j == 0) {
				currImg = imglist[0];
			} else {

				cout << "\n\n\ni, j :" << i << "," << j << endl;
				currImg = getMinSSDImg(prevImg, topImg, hSrc, width, height);
			}

			placeImg(i, j, currImg, hDst);

		}
	}

	//dSrc.copyTo(dDst);
	//dDst.download(hDst);
}

int main() {


	int num_devices = getCudaEnabledDeviceCount();
	cout << "gpu count :" << num_devices << endl;

	std::cout << "Hello World" << std::endl;
	std::string imageName = "image1.png";
	cv::Mat input = cv::imread(imageName, CV_LOAD_IMAGE_COLOR);

	if (input.empty()) {
		cout << "Cannot read " + imageName << endl;
	} else {
		cout << imageName + " loaded" << endl;
		/*cout << input << endl;
		int b = input.at<cv::Vec3b>(0,0)[0];
		int g = input.at<cv::Vec3b>(0,0)[1];
		int r = input.at<cv::Vec3b>(0,0)[2];

		int val = input.at<int>(0,0);
		cout << b << g << r <<  endl;*/
	}
	cv::Mat output(outputY_size, outputX_size, CV_8UC3);

	imageQuilting(input, output);

	//cv::imshow("Output", output);

	cv::waitKey();

	return 0;
}
